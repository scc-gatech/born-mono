/* include this file to catch cuda runtime and cuda device API errors */
/* By: Abdullah AlTheyab (2009) */
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <builtin_types.h>
#define kernel_exec(x,y) x,y; cuda_kernel_error(__FILE__, __LINE__)
inline void cuda_kernel_error(char* file, int linenum){
	hipError_t errcode=hipGetLastError();
	if(errcode!=hipSuccess){
		printf("Kernel error in file %s line %d: %s\n", file, linenum, hipGetErrorString(errcode));
		exit(-1);
	}
}

#define cuda_call(x) cuda_call_check(__FILE__, __LINE__, x)
inline void cuda_call_check(char* file, int linenum, hipError_t errcode){
	if(errcode!=hipSuccess){
		printf("CUDA error in file %s line %d: %s\n", file, linenum, hipGetErrorString(errcode));
		exit(-1);
	}
}

#define cu_call(x) cu_call_check(__FILE__, __LINE__, x)
inline void cu_call_check(char* file, int linenum, hipError_t status){
	if(status!=hipSuccess){
		char * msg;
		
		switch(status){
			case hipErrorDeinitialized:
				msg="CUDA_ERROR_DEINITIALIZED"; break;
			case hipErrorNotInitialized:
				msg="CUDA_ERROR_NOT_INITIALIZED"; break;
			case hipErrorInvalidContext:
				msg="CUDA_ERROR_INVALID_CONTEXT"; break;
			case hipErrorInvalidHandle:
				msg="CUDA_ERROR_INVALID_HANDLE"; break;
			case hipErrorInvalidDevice:
				msg="CUDA_ERROR_INVALID_DEVICE"; break;
			case hipErrorInvalidValue:
				msg="CUDA_ERROR_INVALID_VALUE"; break;
			case CUDA_ERROR_LAUNCH_INCOMPATIBLE_TEXTURING:
				msg="CUDA_ERROR_LAUNCH_INCOMPATIBLE_TEXTURING"; break;
			case hipErrorOutOfMemory:
				msg="CUDA_ERROR_OUT_OF_MEMORY"; break;
			case hipErrorNotFound:
				msg="CUDA_ERROR_NOT_FOUND"; break;
			case hipErrorFileNotFound:
				msg="CUDA_ERROR_FILE_NOT_FOUND"; break;
			case hipErrorNoBinaryForGpu:
				msg="CUDA_ERROR_NO_BINARY_FOR_GPU"; break;
			case hipErrorNotReady:
				msg="CUDA_ERROR_NOT_READY"; break;
			case hipErrorNoDevice:
				msg="CUDA_ERROR_NO_DEVICE"; break;
			case hipErrorArrayIsMapped:
				msg="CUDA_ERROR_ARRAY_IS_MAPPED"; break;
			case hipErrorMapFailed:
				msg="CUDA_ERROR_MAP_FAILED"; break;
			case hipErrorAlreadyMapped:
				msg="CUDA_ERROR_ALREADY_MAPPED"; break;
			case hipErrorNotMapped:
				msg="CUDA_ERROR_NOT_MAPPED"; break;
			case hipErrorUnknown:
				msg="CUDA_ERROR_UNKNOWN"; break;
			default:
				msg="I don't know!!"; break;
		}
		fprintf(stderr, "*Cuda driver API error code %d in %s:%d: %s.\n", status, file, linenum, msg );
		exit(-1);
	}
}
