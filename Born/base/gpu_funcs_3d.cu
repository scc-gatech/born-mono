#include "hip/hip_runtime.h"
//#include "cutil_inline.h"
//#include "cutil_math.h"
#include "gpu_finite_3d.h"
#include <stdlib.h>
#include "sep3d.h"
#include "seplib.h"  
#include "cudaErrors.cu"
int ntblock_internal;
#include "wave_fkernel.3d8o.cu"
#include"assert.h"

float *source_buf;
int npts_internal,source_blocked,ntsource_internal;


void setup_cuda(int ngpus, int argc, char **argv){
  n_gpus=ngpus;
  fprintf(stderr,"Today, we are using %d GPUs; specifically: \n",n_gpus);
  int dr;

  for(int i=0; i<n_gpus; i++) device[i]=i;

  for(int i=0; i<n_gpus; i++){
    hipDeviceSynchronize();

    hipSetDevice(device[i]);
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties,device[i]);
    hipDriverGetVersion(&dr);

    fprintf(stderr,"  GPU %s (%d),",properties.name, device[i]);
    if(properties.unifiedAddressing) fprintf(stderr," UVA initiated,");
    else fprintf(stderr," UVA not working ");
    fprintf(stderr," driver %d\n",dr);

    //Enable P2P memcopies between GPUs
    if(n_gpus > 1){
      for(int j=0; j<n_gpus; j++){
        if(i==j) continue;
        int peer_access_available=0;
        hipDeviceCanAccessPeer( &peer_access_available,device[i],device[j]);
        if(peer_access_available){
	  //fprintf(stderr,"Make the GPUs talk %d %d\n",device[i],device[j]);
          hipDeviceEnablePeerAccess(device[j],0);
        }
      }
    }

  }

}

void process_error( const hipError_t &error, char *string=0, bool verbose=false ){
    if( error != hipSuccess || verbose )
    {
        int current_gpu = -1;
        hipGetDevice( &current_gpu );

        fprintf(stderr, "GPU %d: ", current_gpu );
        if( string )
            printf( string );
        fprintf(stderr, ": %s\n", hipGetErrorString( error ) );
    }

    if( error != hipSuccess )
        exit(-1);
}

extern "C" __global__ void new_src_inject_kernel(int it, int isinc,float *p){
  int ix=blockIdx.x*blockDim.x+threadIdx.x;
  p[srcgeom_gpu0[ix]]+=dir_gpu*(
  sinc_s_table[isinc*nsinc_gpu]*  source_gpu0[ntblock_gpu*ix+it]+
  sinc_s_table[isinc*nsinc_gpu+1]*source_gpu0[ntblock_gpu*ix+it+1]+
  sinc_s_table[isinc*nsinc_gpu+2]*source_gpu0[ntblock_gpu*ix+it+2]+
  sinc_s_table[isinc*nsinc_gpu+3]*source_gpu0[ntblock_gpu*ix+it+3]+
  sinc_s_table[isinc*nsinc_gpu+4]*source_gpu0[ntblock_gpu*ix+it+4]+
  sinc_s_table[isinc*nsinc_gpu+5]*source_gpu0[ntblock_gpu*ix+it+5]+
  sinc_s_table[isinc*nsinc_gpu+6]*source_gpu0[ntblock_gpu*ix+it+6]+
  sinc_s_table[isinc*nsinc_gpu+7]*source_gpu0[ntblock_gpu*ix+it+7]);
}
extern "C" __global__ void new_src_inject2_kernel(int it, int isinc,float *p,float *source_gpu1){
  int j=blockIdx.y*blockDim.y+threadIdx.y;
  int k=blockIdx.x*blockDim.x+threadIdx.x;
  int i=k+n1gpu*j;
  if(i<rec_nx_gpu*rec_ny_gpu){
    p[srcgeom_gpu0[i]]+=
      dir_gpu*(
      sinc_s_table[isinc*nsinc_gpu]*  source_gpu0[ntblock_gpu*i+it]+
      sinc_s_table[isinc*nsinc_gpu+1]*source_gpu0[ntblock_gpu*i+it+1]+
      sinc_s_table[isinc*nsinc_gpu+2]*source_gpu0[ntblock_gpu*i+it+2]+
      sinc_s_table[isinc*nsinc_gpu+3]*source_gpu0[ntblock_gpu*i+it+3] +
      sinc_s_table[isinc*nsinc_gpu+4]*source_gpu0[ntblock_gpu*i+it+4]+
      sinc_s_table[isinc*nsinc_gpu+5]*source_gpu0[ntblock_gpu*i+it+5]+
      sinc_s_table[isinc*nsinc_gpu+6]*source_gpu0[ntblock_gpu*i+it+6]+
      sinc_s_table[isinc*nsinc_gpu+7]*source_gpu0[ntblock_gpu*i+it+7]
    
      );
    
  }
}
extern "C" __global__ void new_data_inject_kernel(int it, int isinc,float *p){
  int j=blockIdx.y*blockDim.y+threadIdx.y;
  int k=blockIdx.x*blockDim.x+threadIdx.x;
  int i=k+n1gpu*j;
  if(i< rec_nx_gpu*rec_ny_gpu){
    p[datageom_gpu0[i]]+=
    sinc_d_table[isinc*nsinc_gpu]*data_gpu0[ntrace_gpu*i+it] +
    sinc_d_table[isinc*nsinc_gpu+1]*data_gpu0[ntrace_gpu*i+it+1]+
    sinc_d_table[isinc*nsinc_gpu+2]*data_gpu0[ntrace_gpu*i+it+2]+
    sinc_d_table[isinc*nsinc_gpu+3]*data_gpu0[ntrace_gpu*i+it+3]+
    sinc_d_table[isinc*nsinc_gpu+4]*data_gpu0[ntrace_gpu*i+it+4]+
    sinc_d_table[isinc*nsinc_gpu+5]*data_gpu0[ntrace_gpu*i+it+5]+
    sinc_d_table[isinc*nsinc_gpu+6]*data_gpu0[ntrace_gpu*i+it+6]+
    sinc_d_table[isinc*nsinc_gpu+7]*data_gpu0[ntrace_gpu*i+it+7];
  }

}

extern "C" __global__ void zero_data(float *p){
  long long j=blockIdx.y*blockDim.y+threadIdx.y;
  long long k=blockIdx.x*blockDim.x+threadIdx.x;
  long long i=k+(n1gpu)*j;
  long long it;
  long long nt=ntblock_gpu;

   if(i< (rec_nx_gpu*rec_ny_gpu)){
     for(it=0; it < ntblock_gpu; it++){
      data_gpu0[nt*i+it]=0;
    }
  }
}
extern "C" __global__ void move_zero_data(float *p){
  long long j=blockIdx.y*blockDim.y+threadIdx.y;
  long long k=blockIdx.x*blockDim.x+threadIdx.x;
  long long i=k+(n1gpu)*j;
  long long it;
  long long nt=ntblock_gpu-7;

  if(i< (rec_nx_gpu*rec_ny_gpu)){
     for(it=0; it < 7; it++)           data_gpu0[ntblock_gpu*i+it]=data_gpu0[ntblock_gpu*i+it+nt];
     for(it=7; it < ntblock_gpu; it++) data_gpu0[ntblock_gpu*i+it]=0;
    
 }
}
extern "C" __global__ void new_data_extract_kernel(int it, int isinc,float *p){
  int j=blockIdx.y*blockDim.y+threadIdx.y;
  int k=blockIdx.x*blockDim.x+threadIdx.x;
  int i=k+(n1gpu)*j;
  

  if(i< (rec_nx_gpu*rec_ny_gpu)){
  
   data_gpu0[ntblock_gpu*(i)+it+0]+=p[datageom_gpu0[i]]*sinc_d_table[isinc*nsinc_gpu+0];
    data_gpu0[ntblock_gpu*(i)+it+1]+=p[datageom_gpu0[i]]*sinc_d_table[isinc*nsinc_gpu+1];
    data_gpu0[ntblock_gpu*(i)+it+2]+=p[datageom_gpu0[i]]*sinc_d_table[isinc*nsinc_gpu+2];
    data_gpu0[ntblock_gpu*(i)+it+3]+=p[datageom_gpu0[i]]*sinc_d_table[isinc*nsinc_gpu+3];
    data_gpu0[ntblock_gpu*(i)+it+4]+=p[datageom_gpu0[i]]*sinc_d_table[isinc*nsinc_gpu+4];
    data_gpu0[ntblock_gpu*(i)+it+5]+=p[datageom_gpu0[i]]*sinc_d_table[isinc*nsinc_gpu+5];
    data_gpu0[ntblock_gpu*(i)+it+6]+=p[datageom_gpu0[i]]*sinc_d_table[isinc*nsinc_gpu+6];
    data_gpu0[ntblock_gpu*(i)+it+7]+=p[datageom_gpu0[i]]*sinc_d_table[isinc*nsinc_gpu+7];
    
    }
   

}
extern "C" __global__ void img_kernel( float* img, float*dat, float*src){
  int ig = blockIdx.x * blockDim.x + threadIdx.x;
  int jg = blockIdx.y * blockDim.y + threadIdx.y;
  int addr= ig + n1gpu * jg;
  int stride = n1gpu*n2gpu;
  for(int iy=0; iy<n3gpu; iy++){
    img[addr]+=.000001*dat[addr]*src[addr];
    addr+=stride;
  }
}
extern "C" __global__ void img_add_kernel( float* img, float*rec_field, float *src_field){//as above, added 
 long long ig = blockIdx.x * blockDim.x + threadIdx.x;
  long long jg = blockIdx.y * blockDim.y + threadIdx.y;
  long long addr =  ig + (long long) n1gpu * jg;
  long long  stride = (long long) n1gpu*(long long)n2gpu;
  for(long long iy=0; iy<n3gpu; iy++){
    rec_field[addr]+=.000001*src_field[addr]*(img[addr]);
    addr+=stride;
  }
}
void source_prop(int n1, int n2, int n3, bool damp, bool get_last, float *p0, float *p1, int jt, int npts,int nt){

  //Propagate the source wavefield and return the final two 3D wavefield slices

  float *ptemp;
  float *src_p0[n_gpus],*src_p1[n_gpus];

  hipError_t error = hipSuccess;

  //int n3_total=n3;
  n3=(n3-2*radius)/n_gpus + 2*radius;
  //int dim3=n3;
  //if(n_gpus > 1) dim3-=2*radius;

  int dir=1;

  int n_bytes_gpu=(n1*n2*n3+lead_pad)*sizeof(float);

  for(int i=0; i<n_gpus; i++){

    hipSetDevice(device[i]);
    //hipDeviceSetCacheConfig( hipFuncCachePreferL1 );

    hipMalloc((void**) &src_p0[i],n_bytes_gpu);
    hipMalloc((void**) &src_p1[i],n_bytes_gpu);

    hipMemset(src_p0[i], 0,n_bytes_gpu);
    hipMemset(src_p1[i], 0,n_bytes_gpu);

    hipMemcpyToSymbol(HIP_SYMBOL(dir_gpu), &dir, sizeof(float));

  }

//fprintf(stderr,"Allocate %d %d %d, %f mbs; %d\n",n1,n2,n3,(float)(n1*n2*n3*4/1000000),lead_pad);

  //Blocks for internal data
  //int nblocks1=(n1-2*FAT)/(2*BLOCKX_SIZE);
  int nblocks1=(n1-2*FAT)/BLOCKX_SIZE;
  int nblocks2=(n2-2*FAT)/BLOCKY_SIZE;

  dim3 dimBlock(BLOCKX_SIZE, BLOCKY_SIZE);
  dim3 dimGrid(nblocks1,nblocks2);

  //Define separate streams for overlapping communication
  hipStream_t stream_halo[n_gpus], stream_internal[n_gpus];
  hipEvent_t start,stop;

  hipSetDevice(device[0]);
  hipEventCreate(&start);
  hipEventCreate(&stop); 

  //Setup coordinate systems for internal domains
  int offset_internal[n_gpus];
  int start3[n_gpus],end3[n_gpus];

  for(int i=0; i<n_gpus; i++){
    hipSetDevice(device[i]);
    hipStreamCreate(&stream_halo[i]);
    hipStreamCreate(&stream_internal[i]);

    //Offset_internal is the initial index of our internal domain (out of stencil padding)
    offset_internal[i]=offset;
    if(i > 0) offset_internal[i] += n1*n2*radius;

    start3[i] = i*(n3-2*radius) + 2*radius;
    end3[i] = (i+1)*(n3-2*radius) /*- radius*/;
    //start3[i] = i*(n3-2*radius) + radius;
    //end3[i] = (i+1)*(n3-2*radius) - radius;
  }

  start3[0]=radius;
  end3[n_gpus-1]=n_gpus*(n3-2*radius);//I THINK THIS SHOULD BE -RADIUS. LET'S TRY
  //start3[0]=0;
  //end3[n_gpus-1]=n_gpus*(n3-2*radius);

  //Set up coordinate systems for the halo exchange
  int offset_snd_h1=lead_pad+n1*n2*radius;
  int offset_snd_h2=lead_pad+n1*n2*(n3-2*radius);
  int offset_rcv_h1=lead_pad;
  int offset_rcv_h2=lead_pad+n1*n2*(n3-2*radius+radius);
  int offset_cmp_h1=offset;
  long int offset_cmp_h2=lead_pad+radius+radius*n1+n1*n2*(n3-2*radius);//-radius?

  /*int offset_snd_h1=lead_pad+n1*n2*radius;
  int offset_snd_h2=lead_pad+n1*n2*(n3-2*radius);
  int offset_rcv_h1=lead_pad;
  int offset_rcv_h2=lead_pad+n1*n2*(n3-2*radius+radius);
  int offset_cmp_h1=offset;
  long int offset_cmp_h2=lead_pad+radius+radius*n1+radius*n1*n2+n1*n2*(n3-2*radius-radius);*/


  hipSetDevice(device[0]);
  hipEventRecord(start,0);

  for(int it=0; it<=nt; it++){
    int id=it/jt;
    int ii=it-id*jt;

    // Calculate the halo regions first
    for(int i=0; i<n_gpus; i++){
      hipSetDevice(i);

      if(i>0){
        wave_kernel<<<dimGrid,dimBlock,0,stream_halo[i]>>>(src_p0[i]+offset_cmp_h1, src_p1[i]+offset_cmp_h1, src_p0[i]+offset_cmp_h1, velocity[i]+offset_cmp_h1, radius, 2*radius);
      }

      if(i<n_gpus-1){
        wave_kernel<<<dimGrid,dimBlock,0,stream_halo[i]>>>(src_p0[i]+offset_cmp_h2, src_p1[i]+offset_cmp_h2, src_p0[i]+offset_cmp_h2, velocity[i]+offset_cmp_h2, (n3-radius)-radius,n3-radius);
      }

      hipStreamQuery(stream_halo[i]);
    }

    for(int i=0; i<n_gpus; i++){
      hipSetDevice(device[i]);

      wave_kernel<<<dimGrid, dimBlock,0,stream_internal[i]>>>(src_p0[i]+offset_internal[i], src_p1[i]+offset_internal[i], src_p0[i]+offset_internal[i], velocity[i]+offset_internal[i], start3[i], end3[i]);
      if(i==shot_gpu) new_src_inject_kernel<<<1,npts,0,stream_internal[i]>>>(id ,ii, src_p0[i]+lead_pad);
    }

    //Overlap internal computation with halo communication

    //Send halos to the 'right'
    for(int i=0; i<n_gpus-1; i++){
      hipMemcpyPeerAsync(src_p0[i+1]+offset_rcv_h1,i+1,src_p0[i]+offset_snd_h2,i,n1*n2*radius*sizeof(float),stream_halo[i]);
    }

    //Synchronize to avoid stalling
    for(int i=0; i<n_gpus-1; i++){
      hipSetDevice(i);
      hipStreamSynchronize(stream_halo[i]);
    }

    //Send halos to the 'left'
    for(int i=1; i<n_gpus; i++){
      hipMemcpyPeerAsync(src_p0[i-1]+offset_rcv_h2,i-1,src_p0[i]+offset_snd_h1,i,n1*n2*radius*sizeof(float),stream_halo[i]);
    }

    //Synchronise GPUs and do pointer exchange
    for(int i=0; i<n_gpus; i++){
      hipSetDevice(device[i]);
      hipDeviceSynchronize();
      ptemp=src_p0[i]; src_p0[i]=src_p1[i];src_p1[i]=ptemp;
    }
  }

  error = hipGetLastError();
  process_error( error, "kernel" );

  //Use device 0 to give a performance report
  hipSetDevice(device[0]);
  hipEventRecord(stop,0);

  hipDeviceSynchronize();

  float time_total;
  hipEventElapsedTime(&time_total,start,stop);
  fprintf(stderr,"Time for source propagation = %f seconds \n",time_total/1000);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  if(get_last){
    for(int i=0; i<n_gpus; i++){
      hipSetDevice(device[i]);
      hipMemcpy(p0+i*n1*n2*(n3-2*radius), src_p0[i]+lead_pad/*+radius*n1*n2*/, n1*n2*(n3/*-2*radius*/)*sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(p1+i*n1*n2*(n3-2*radius), src_p1[i]+lead_pad/*+radius*n1*n2*/, n1*n2*(n3/*-2*radius*/)*sizeof(float), hipMemcpyDeviceToHost);
      //hipMemcpy(p0+i*n1*n2*(n3-radius), src_p0[i]+radius*n1*n2, n1*n2*(n3-radius)*sizeof(float), hipMemcpyDeviceToHost);
      //hipMemcpy(p1+i*n1*n2*(n3-radius), src_p1[i]+radius*n1*n2, n1*n2*(n3-radius)*sizeof(float), hipMemcpyDeviceToHost);
    //hipMemcpy(src_p0[i]+lead_pad/*+radius*n1*n2*/, p0_s_cpu+i*n1*n2*(n3-2*radius), n1*n2*(n3/*-2*radius*/)*sizeof(float),
    }
  }


  for(int i=0; i<n_gpus; i++){
    hipSetDevice(device[i]);
    hipFree(src_p0[i]);
    hipFree(src_p1[i]);
  }
  hipFree(source_gpu);
  hipFree(srcgeom_gpu);

}
void rtm_forward(int n1, int n2, int n3, int jt, float *img, float *dat, int npts_src, int nt,int nt_big, int rec_nx, int rec_ny){



if(1==1){

  //Born modelling over input image
  float *src_p0[n_gpus], *src_p1[n_gpus], *data_p0[n_gpus], *data_p1[n_gpus], *img_gpu[n_gpus];
  float *ptemp, *ptemp2;

  hipError_t error = hipSuccess;

  int n3_total=n3;
  n3=(n3-2*radius)/n_gpus + 2*radius;

  int dir=1;

  int n3s=n3-2*radius;

  int nblocks1=(n1-2*FAT)/BLOCKZ_SIZE; 
  int nblocks2=(n2-2*FAT)/BLOCKX_SIZE; 
  //int nblocks3=(n3-2*FAT)/BLOCKY_SIZE; 

  dim3 dimGrid(nblocks1,nblocks2);
  dim3 dimBlock(16, 16);
  
  //dim3 dimGridx((int)ceilf(1.*n1/BLOCKX_SIZE),(int)ceilf(1.*n2/BLOCKY_SIZE));
  dim3 dimGridx((int)ceilf(1.*n1/BLOCKX_SIZE),(int)ceilf(1.*n2/BLOCKY_SIZE));
fprintf(stderr,"CEHCK GRID %d %d %d %d  \n",dimBlock.x,dimBlock.y,dimGridx.x,dimGridx.y);



  hipStream_t stream_halo[n_gpus], stream_internal[n_gpus];
  hipEvent_t start,stop;

  hipSetDevice(device[0]);
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int offset_internal[n_gpus];
  int start3[n_gpus],end3[n_gpus];

  for(int i=0; i<n_gpus; i++){
    hipSetDevice(device[i]);
    hipStreamCreate(&stream_halo[i]);
    hipStreamCreate(&stream_internal[i]);

    offset_internal[i]=offset;
    if(i > 0) offset_internal[i] += n1*n2*radius;

    start3[i] = i*(n3-2*radius) + 2*radius;
    end3[i] = (i+1)*(n3-2*radius);
  }

  start3[0]=radius;
  end3[n_gpus-1]=n_gpus*(n3-2*radius);

  for(int i=0; i<n_gpus; i++){

    hipSetDevice(device[i]);
    hipMalloc((void**) &src_p0[i], (n1*n2*n3+lead_pad)*sizeof(float));
    hipMalloc((void**) &src_p1[i], (n1*n2*n3+lead_pad)*sizeof(float));
    hipMalloc((void**) &data_p0[i],(n1*n2*n3+lead_pad)*sizeof(float));
    hipMalloc((void**) &data_p1[i],(n1*n2*n3+lead_pad)*sizeof(float));
    hipMalloc((void**) &img_gpu[i], n1*n2*n3*sizeof(float));

    hipMemset(data_p0[i], 0,(n1*n2*n3+lead_pad)*sizeof(float));
    hipMemset(data_p1[i], 0,(n1*n2*n3+lead_pad)*sizeof(float));
    hipMemset(src_p0[i],  0,(n1*n2*n3+lead_pad)*sizeof(float));
    hipMemset(src_p1[i],  0,(n1*n2*n3+lead_pad)*sizeof(float));
    hipMemset(img_gpu[i], 0,n1*n2*n3*sizeof(float));

    hipMemcpy( img_gpu[i]/*+radius*n1*n2*/, img+i*n1*n2*(n3-2*radius), n1*n2*(n3/*-2*radius*/)*sizeof(float), hipMemcpyHostToDevice);
    //DONT DELETE need the -2*r gone to be multi-gpu invariant
    //hipMemcpy( img_gpu[i]+radius*n1*n2, img+i*n1*n2*(n3-2*radius), n1*n2*(n3-2*radius)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(dir_gpu), &dir, sizeof(float));
  }

  int offset_snd_h1=lead_pad+n1*n2*radius;
  int offset_snd_h2=lead_pad+n1*n2*(n3-2*radius);
  int offset_rcv_h1=lead_pad+0;
  int offset_rcv_h2=lead_pad+n1*n2*(n3-2*radius+radius);

  int offset_cmp_h1=offset;
  int offset_cmp_h2=lead_pad+radius+radius*n1+n1*n2*(n3-radius-radius);

  hipSetDevice(device[0]);
  hipEventRecord(start,0);
  long long iblock=0;
  int icycle=-1;
  //zero_data<<<dimGridx,dimBlock,0,stream_internal[0]>>>(data_p0[0]);
/*
 float *temp=(float*)malloc(sizeof(float)*rec_nx*rec_ny*(ntblock_internal+7));
      hipMemcpy(temp, data_gpu, (7+ntblock_internal)*rec_nx*rec_ny*sizeof(float), hipMemcpyDeviceToHost);
     srite("book.H",temp,(7+ntblock_internal)*rec_nx*rec_ny*sizeof(float));
free(temp);
*/

  for(int it=0; it < nt; it++){
    int id=it/jt;
    int ii=it-id*jt;
    int id_block=id-((int)(id/ntblock_internal))*ntblock_internal;
    if(it%100==10){
  //   fprintf(stderr,"WRITING WAVEFIELD %d %d %d \n",n1,n2,n3_total);
  //   writeWavefield("src.H",src_p0,n3s,n_gpus,n1,n2,n3_total,radius);
   //       writeWavefield("dat.H",data_p0,n3s,n_gpus,n1,n2,n3_total,radius);

    
    
    }

    for(int i=0; i<n_gpus; i++){
      hipSetDevice(device[i]);
      if(i>0){
        wave_kernel_adj<<<dimGrid,dimBlock,0,stream_halo[i]>>>(data_p0[i]+offset_cmp_h1, data_p1[i]+offset_cmp_h1, data_p0[i]+offset_cmp_h1, velocity[i]+offset_cmp_h1, radius, 2*radius);
        wave_kernel<<<dimGrid,dimBlock,0,stream_halo[i]>>>(src_p0[i]+offset_cmp_h1, src_p1[i]+offset_cmp_h1, src_p0[i]+offset_cmp_h1, velocity[i]+offset_cmp_h1, radius, 2*radius);
      } 
      if(i<n_gpus-1){
        wave_kernel_adj<<<dimGrid,dimBlock,0,stream_halo[i]>>>(data_p0[i]+offset_cmp_h2, data_p1[i]+offset_cmp_h2, data_p0[i]+offset_cmp_h2, velocity[i]+offset_cmp_h2, (n3-radius)-radius, n3-radius);
        wave_kernel<<<dimGrid,dimBlock,0,stream_halo[i]>>>(src_p0[i]+offset_cmp_h2, src_p1[i]+offset_cmp_h2, src_p0[i]+offset_cmp_h2, velocity[i]+offset_cmp_h2, (n3-radius)-radius, n3-radius);
      }
      hipStreamQuery(stream_halo[i]);
    }
    
    if(ii==0) {
      icycle++;
      if(icycle%ntblock_internal==0){
        load_source(id);
      }
    }

    for(int i=0; i<n_gpus; i++){
      hipSetDevice(device[i]);

      damp_kernel<<<dimGrid,dimBlock,0,stream_internal[i]>>>(data_p0[i], data_p1[i], start3[i], end3[i], i, n_gpus);
      damp_kernel<<<dimGrid,dimBlock,0,stream_internal[i]>>>(data_p0[i], data_p1[i], start3[i], end3[i], i, n_gpus);
      damp_kernel<<<dimGrid,dimBlock,0,stream_internal[i]>>>(src_p0[i], src_p1[i], start3[i], end3[i], i, n_gpus);

      wave_kernel_adj<<<dimGrid,dimBlock,0,stream_internal[i]>>>(data_p0[i]+offset_internal[i], data_p1[i]+offset_internal[i], data_p0[i]+offset_internal[i], velocity[i]+offset_internal[i], start3[i], end3[i]);

      wave_kernel<<<dimGrid,dimBlock,0,stream_internal[i]>>>(src_p0[i]+offset_internal[i], src_p1[i]+offset_internal[i], src_p0[i]+offset_internal[i], velocity[i]+offset_internal[i], start3[i], end3[i]);

    fprintf(stderr,"in m0odeling %d of %d %d \n",it,nt,id_block);

      if(i==shot_gpu) {
      
         if(npts_src<100) new_src_inject_kernel<<<1,npts_src,0,stream_internal[i]>>>(icycle ,ii,src_p0[i]+lead_pad);
         
         else{
           new_src_inject2_kernel              <<<dimGridx,dimBlock,0,stream_internal[i]>>>(icycle ,ii,src_p0[i]+lead_pad,source_gpu);
         }
      }
    }
    
if(1==3){
 float *temp=(float*)malloc(sizeof(float)*rec_nx*rec_ny*(ntblock_internal+7));
      hipMemcpy(temp, data_gpu, (7+ntblock_internal)*rec_nx*rec_ny*sizeof(float), hipMemcpyDeviceToHost);
      srite("book.H",temp,(7+ntblock_internal)*rec_nx*rec_ny*sizeof(float));
free(temp);
}
    for(int i=0; i<n_gpus-1; i++){
      hipMemcpyPeerAsync(data_p0[i+1]+offset_rcv_h1,i+1,data_p0[i]+offset_snd_h2,i,n1*n2*radius*sizeof(float),stream_halo[i]);
      hipMemcpyPeerAsync(src_p0[i+1]+offset_rcv_h1,i+1,src_p0[i]+offset_snd_h2,i,n1*n2*radius*sizeof(float),stream_halo[i]);
    }
    for(int i=0; i<n_gpus-1; i++){
      hipSetDevice(device[i]);
      hipStreamSynchronize(stream_halo[i]);
    }
    for(int i=1; i<n_gpus; i++){
      hipMemcpyPeerAsync(data_p0[i-1]+offset_rcv_h2,i-1,data_p0[i]+offset_snd_h1,i,n1*n2*radius*sizeof(float),stream_halo[i]);
      hipMemcpyPeerAsync(src_p0[i-1]+offset_rcv_h2,i-1,src_p0[i]+offset_snd_h1,i,n1*n2*radius*sizeof(float),stream_halo[i]);
    }
    for(int i=0; i<n_gpus; i++){ 
      hipSetDevice(device[i]);
      hipSetDevice(device[i]);
      hipStreamSynchronize(stream_internal[i]);
    }
    for(int i=0; i<n_gpus; i++){
      hipSetDevice(device[i]);
      //if(it%jt==0) img_add_kernel<<<dimGrid,dimBlock,0,stream_internal[i]>>>(img_gpu[i]+offset_snd_h1,data_p0[i]+offset_snd_h1,src_p0[i]+offset_snd_h1);
      if(it%jt==0) img_add_kernel<<<dimGrid,dimBlock,0,stream_internal[i]>>>(img_gpu[i],data_p0[i],src_p0[i]);
      //if(it%jt==0) img_add_kernel<<<dimGrid,dimBlock,0,stream_internal[i]>>>(img_gpu[i]+offset_internal[i]-offset,data_p0[i]+offset_internal[i]-offset,src_p0[i]+offset_internal[i]-offset); //works for multis
    }
    fprintf(stderr,"in 2modeling %d of %d %d \n",it,nt,id_block);

    for(int i=0; i<n_gpus; i++){
      hipSetDevice(device[i]);
      hipStreamSynchronize(stream_internal[i]);
      hipDeviceSynchronize();
    }

    hipSetDevice(device[0]);

    if(icycle==ntblock_internal){
      icycle=0;
   
      float *temp=(float*)malloc(sizeof(float)*rec_nx*rec_ny*(ntblock_internal+7));
      hipMemcpy(temp, data_gpu, (7+ntblock_internal)*rec_nx*rec_ny*sizeof(float), hipMemcpyDeviceToHost);
      srite("book.H",temp,(7+ntblock_internal)*rec_nx*rec_ny*sizeof(float));
    float sm=0;
      for(int k=0; k <rec_nx*rec_ny*ntblock_internal; k++){
        sm+=fabs(temp[k]);
      }
      int itr=0;
      for(int iy=0; iy < rec_ny; iy++){
        for(int ix=0; ix < rec_nx; ix++,itr++){
           memcpy(&dat[nt_big*itr+iblock*ntblock_internal],&temp[(ntblock_internal+7)*itr],ntblock_internal*sizeof(float));
        }
      }
      iblock++;
      move_zero_data<<<dimGridx,dimBlock,0,stream_internal[0]>>>(data_p0[0]);
    //  hipMemcpy(temp, data_gpu, (7+ntblock_internal)*rec_nx*rec_ny*sizeof(float), hipMemcpyDeviceToHost);
    //  srite("book.H",temp,(7+ntblock_internal)*rec_nx*rec_ny*sizeof(float));

      free(temp);
    }
    new_data_extract_kernel<<<dimGridx,dimBlock,0,stream_internal[0]>>>(icycle ,ii,data_p0[0]/*,datageom_gpu,data_gpu+offset_snd_h1*/);
    fprintf(stderr,"in modelaxing %d of %d %d \n",it,nt,id_block);

    for(int i=0; i<n_gpus; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
      ptemp2=src_p1[i]; src_p1[i]=src_p0[i]; src_p0[i]=ptemp2;
      ptemp=data_p1[i]; data_p1[i]=data_p0[i]; data_p0[i]=ptemp;
    }
   /*
    fprintf(stderr,"in modelin2g %d of %d %d \n",it,nt,id_block);
 float *temp=(float*)malloc(sizeof(float)*rec_nx*rec_ny*(ntblock_internal+7));
      hipMemcpy(temp, data_gpu, (7+ntblock_internal)*rec_nx*rec_ny*sizeof(float), hipMemcpyDeviceToHost);
      srite("book.H",temp,(7+ntblock_internal)*rec_nx*rec_ny*sizeof(float));
free(temp);

     writeWavefield("src.H",src_p0,n3s,n_gpus,n1,n2,n3_total,radius);
    seperr("");
    */
  }

  //error = hipGetLastError();
  //process_error( error, "kernel" );
  int ic=0;
  if(nt_big!=ntblock_internal*iblock){
     float *temp=(float*)malloc(sizeof(float)*rec_nx*rec_ny*(ntblock_internal+7));
     hipMemcpy(temp, data_gpu, 
       (7+ntblock_internal)*rec_nx*rec_ny*sizeof(float), 
         hipMemcpyDeviceToHost);
   
     int itr=0;
     for(int iy=0; iy < rec_ny; iy++){
        for(int ix=0; ix < rec_nx; ix++,itr++){
          memcpy(&dat[nt_big*itr+iblock*ntblock_internal],
           &temp[(ntblock_internal+7)*itr],(nt_big-ntblock_internal*iblock)*sizeof(float));
        }
     }

           float sm=0;
      for(int k=0; k <rec_nx*rec_ny*ntblock_internal; k++){
        sm+=fabs(temp[k]);
        if(fabs(temp[k])>.000001) ic++;
      }
     free(temp);
   }
   
   
  hipSetDevice(device[0]);
  hipEventRecord(stop,0);

  hipDeviceSynchronize();

  float time_total;
  hipEventElapsedTime(&time_total,start,stop);
  fprintf(stderr,"Time for Born modelling = %f seconds \n",time_total/1000);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipSetDevice(device[0]);
 // hipMemcpy(dat, data_gpu, nt_big*rec_nx*rec_ny*sizeof(float), hipMemcpyDeviceToHost);
  for(int i=0; i<n_gpus; i++){

    hipSetDevice(i);

    hipFree(data_p0[i]);


    hipFree(img_gpu[i]);

    hipFree(data_p1[i]);

    hipFree(src_p0[i]);

    hipFree(src_p1[i]);


  }
  hipSetDevice(0);

  hipFree(data_gpu);

  hipFree(datageom_gpu);

  hipFree(source_gpu);

  hipFree(srcgeom_gpu);


}




}
void writeWavefield(char *tag, float **dat, int n3s,int ngpu, int n1, int n2, int n3,int edge){
  long long big_block;
  int igpu,block;
  long long doing,done,toDo;
  long long big=500*1000*1000;
  big_block=(long long)n1*(long long)n2 *(long long)n3;
  big_block=big_block/ngpu+n1*n2;

fprintf(stderr,"CXXX %d %d %d \n",n1,n2,n3s*2);
  float *buf=(float*)malloc(sizeof(float*)*big_block);
  for(igpu=0; igpu < 2; igpu++){
      toDo=(long long)n1*(long long)n2*n3s;
     hipMemcpy(buf,dat[igpu]+n1*n2*edge,toDo*sizeof(float),hipMemcpyDeviceToHost);
     done=0;
     while(done < toDo){
       doing=toDo; if(doing > big) doing=big;
       block=doing;
       srite(tag,&buf[done],block*4);
       done+=doing;
     }
   }
   free(buf);
   // seperr("");
}
void rtm_adjoint(int n1, int n2, int n3, int jt, float *p0_s_cpu, float *p1_s_cpu, float *img, int npts_src,int nt){
 
  float *src_p0[n_gpus],*src_p1[n_gpus],*data_p0[n_gpus],*data_p1[n_gpus],*img_gpu[n_gpus];
  float *ptemp,*ptemp2;

  hipError_t error = hipSuccess;

  int n3_total=n3;
  n3=(n3-2*radius)/n_gpus + 2*radius;

  int dir=-1;

  for(int i=0; i<n_gpus; i++){

    hipSetDevice(device[i]);
    hipMalloc((void**) &src_p0[i],  (n1*n2*n3+radius*n1*n2+lead_pad)*sizeof(float));
    hipMalloc((void**) &src_p1[i],  (n1*n2*n3+radius*n1*n2+lead_pad)*sizeof(float));
    hipMalloc((void**) &data_p0[i], (n1*n2*n3+lead_pad)*sizeof(float));
    hipMalloc((void**) &data_p1[i], (n1*n2*n3+lead_pad)*sizeof(float));

    hipMalloc((void**) &img_gpu[i], n1*n2*n3*sizeof(float));

    hipMemset(data_p0[i], 0,(n1*n2*n3+lead_pad)*sizeof(float));
    hipMemset(data_p1[i], 0,(n1*n2*n3+lead_pad)*sizeof(float));
    hipMemset(img_gpu[i], 0,(n1*n2*n3+lead_pad)*sizeof(float));
    hipMemset(src_p0[i], 0,(n1*n2*n3+radius*n1*n2+lead_pad)*sizeof(float));
    hipMemset(src_p1[i], 0,(n1*n2*n3+radius*n1*n2+lead_pad)*sizeof(float));

    hipMemcpy(src_p0[i]+lead_pad/*+radius*n1*n2*/, p0_s_cpu+i*n1*n2*(n3-2*radius), n1*n2*(n3/*-2*radius*/)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(src_p1[i]+lead_pad/*+radius*n1*n2*/, p1_s_cpu+i*n1*n2*(n3-2*radius), n1*n2*(n3/*-2*radius*/)*sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy(src_p0[i]+lead_pad+radius*n1*n2, p0_s_cpu/*+n1*n2*radius*/+i*n1*n2*(n3-2*radius), n1*n2*(n3-2*radius)*sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy(src_p1[i]+lead_pad+radius*n1*n2, p1_s_cpu/*+n1*n2*radius*/+i*n1*n2*(n3-2*radius), n1*n2*(n3-2*radius)*sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy(src_p0[i]+lead_pad+radius*n1*n2, p0_s_cpu/*+n1*n2*radius*/+i*n1*n2*(n3-radius), n1*n2*(n3-radius)*sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy(src_p1[i]+lead_pad+radius*n1*n2, p1_s_cpu/*+n1*n2*radius*/+i*n1*n2*(n3-radius), n1*n2*(n3-radius)*sizeof(float), hipMemcpyHostToDevice);
   }


  hipMemcpyToSymbol(HIP_SYMBOL(dir_gpu), &dir, sizeof(float));

  int nblocks1=(n1-2*FAT)/BLOCKZ_SIZE;
  int nblocks2=(n2-2*FAT)/BLOCKX_SIZE;
  //int nblocks3=(n3-2*FAT)/BLOCKY_SIZE;

  dim3 dimGrid(nblocks1,nblocks2);
  dim3 dimBlock(16, 16);

  dim3 dimGridx((int)ceilf(1.*n1/BLOCKX_SIZE),(int)ceilf(1.*n2/BLOCKY_SIZE));

  hipStream_t stream_halo[n_gpus], stream_internal[n_gpus];
  hipEvent_t start,stop;

  hipSetDevice(device[0]);
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int offset_internal[n_gpus];
  int start3[n_gpus],end3[n_gpus];

  for(int i=0; i<n_gpus; i++){
    hipSetDevice(device[i]);
    hipStreamCreate(&stream_halo[i]);
    hipStreamCreate(&stream_internal[i]);

    offset_internal[i]=offset;  //  offset=radius*n1*n2+radius*n1+radius+lead_pad;
    if(i > 0) offset_internal[i] += n1*n2*radius;

    start3[i] = i*(n3-2*radius) + 2*radius;
    end3[i] = (i+1)*(n3-2*radius);
  }

  start3[0]=radius;
  end3[n_gpus-1]=n_gpus*(n3-2*radius);

  int offset_snd_h1=lead_pad+n1*n2*radius;
  int offset_snd_h2=lead_pad+n1*n2*(n3-2*radius);
  int offset_rcv_h1=lead_pad;
  int offset_rcv_h2=lead_pad+n1*n2*(n3-2*radius+radius)/*+radius+radius*n1*n2*/;
  int offset_cmp_h1=offset;
  int offset_cmp_h2=lead_pad+radius+radius*n1+n1*n2*(n3-radius-radius);

  hipSetDevice(device[0]);
  hipEventRecord(start,0);

  int id_s=(nt+1)/jt;
  int i_s=nt+1-id_s*jt;

  //new_src_inject_kernel<<<1,npts_src>>>(id_s,i_s,src_p0[0]+lead_pad);

  //float *snap;
  //snap=(float*)malloc(4*n1*n2*n3_total);

  for(int it=nt-1; it >=0 ;it--){
    id_s=(it+1)/jt;
    i_s=it+1-id_s*jt;
    int id=it/jt;
    int ii=it-id*jt;

    for(int i=0; i<n_gpus; i++){
      hipSetDevice(i);
      if(i>0){
       wave_kernel<<<dimGrid,dimBlock,0,stream_halo[i]>>>(data_p0[i]+offset_cmp_h1, data_p1[i]+offset_cmp_h1, data_p0[i]+offset_cmp_h1, velocity[i]+offset_cmp_h1, radius, 2*radius);

      if(it<nt-1) wave_kernel<<<dimGrid,dimBlock,0,stream_halo[i]>>>(src_p0[i]+offset_cmp_h1, src_p1[i]+offset_cmp_h1, src_p0[i]+offset_cmp_h1, velocity[i]+offset_cmp_h1, radius, 2*radius);
      }
      if(i<n_gpus-1){
        wave_kernel<<<dimGrid,dimBlock,0,stream_halo[i]>>>(data_p0[i]+offset_cmp_h2, data_p1[i]+offset_cmp_h2, data_p0[i]+offset_cmp_h2, velocity[i]+offset_cmp_h2, radius, 2*radius);

        if(it<nt-1) wave_kernel<<<dimGrid,dimBlock,0,stream_halo[i]>>>(src_p0[i]+offset_cmp_h2, src_p1[i]+offset_cmp_h2, src_p0[i]+offset_cmp_h2, velocity[i]+offset_cmp_h2, radius, 2*radius); //THIS IS THE NAUGHTY ONE
      }
      hipStreamQuery(stream_halo[i]);
    }
    //damp_kernel<<<dimGrid, dimBlock>>>(nblocksz,data_p0,data_p1);

    for(int i=0; i<n_gpus; i++){
      hipSetDevice(i);
      damp_kernel<<<dimGrid,dimBlock,0,stream_internal[i]>>>(data_p0[i], data_p1[i], start3[i], end3[i], i, n_gpus);

      wave_kernel<<<dimGrid,dimBlock,0,stream_internal[i]>>>(data_p0[i]+offset_internal[i], data_p1[i]+offset_internal[i], data_p0[i]+offset_internal[i], velocity[i]+offset_internal[i], start3[i], end3[i]);

      if(it<(nt-1)){

        wave_kernel<<<dimGrid,dimBlock,0,stream_internal[i]>>>(src_p0[i]+offset_internal[i], src_p1[i]+offset_internal[i], src_p0[i]+offset_internal[i], velocity[i]+offset_internal[i], start3[i], end3[i]);

        if(i==shot_gpu) new_src_inject_kernel<<<1,npts_src,0,stream_internal[i]>>>(id_s,i_s, src_p1[i]+lead_pad); //p1??
      }
    }


    for(int i=1; i<n_gpus; i++){
      hipMemcpyPeerAsync(data_p0[i-1]+offset_rcv_h2,i-1,data_p0[i]+offset_snd_h1,i,n1*n2*radius*sizeof(float),stream_halo[i]);
      if(it<nt-1) hipMemcpyPeerAsync(src_p0[i-1] +offset_rcv_h2,i-1,src_p0[i] +offset_snd_h1,i,n1*n2*radius*sizeof(float),stream_halo[i]);
    }
    for(int i=0; i<n_gpus-1; i++){
      hipSetDevice(i);
      hipStreamSynchronize(stream_halo[i]);
    }
    for(int i=0; i<n_gpus-1; i++){
      hipMemcpyPeerAsync(data_p0[i+1]+offset_rcv_h1,i+1,data_p0[i]+offset_snd_h2,i,n1*n2*radius*sizeof(float),stream_halo[i]);
      if(it<nt-1) hipMemcpyPeerAsync(src_p0[i+1] +offset_rcv_h1,i+1,src_p0[i] +offset_snd_h2,i,n1*n2*radius*sizeof(float),stream_halo[i]);
    }

    hipSetDevice(device[0]);
    new_data_inject_kernel<<<dimGridx,dimBlock,0,stream_internal[0]>>>(id,ii, data_p0[0]/*+offset_snd_h1*/);


    if(ii==0){
      for(int i=0; i<n_gpus; i++){
        hipSetDevice(device[i]);
        img_kernel<<<dimGrid,dimBlock,0,stream_internal[i]>>>(img_gpu[i]+lead_pad,data_p0[i]+lead_pad,src_p0[i]+lead_pad);
      }
    }

    for(int i=0; i<n_gpus; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
      if(it<nt-1){ ptemp=src_p0[i]; src_p0[i]=src_p1[i];src_p1[i]=ptemp;}
      ptemp2=data_p1[i]; data_p1[i]=data_p0[i]; data_p0[i]=ptemp2;
    }
  }

  //error = hipGetLastError();
  //process_error( error, "kernel" );

  hipSetDevice(device[0]);
  hipEventRecord(stop,0);

  hipDeviceSynchronize();

  float time_total;
  hipEventElapsedTime(&time_total,start,stop);
  fprintf(stderr,"Time for imaging = %f seconds \n",time_total/1000);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  for(int i=0; i<n_gpus; i++){
    hipSetDevice(i);
   fprintf(stderr,"CHECK2 %d %d %d \n",n1,n2,n3);
    hipMemcpy(img+i*n1*n2*(n3-2*radius), img_gpu[i]/*+radius*n1*n2*/, n1*n2*(n3/*-2*radius*/)*sizeof(float), hipMemcpyDeviceToHost);
    //hipMemcpy(img+i*n1*n2*(n3-2*radius), img_gpu[i]+radius*n1*n2, n1*n2*(n3-2*radius)*sizeof(float), hipMemcpyDeviceToHost);
    //hipMemcpy( img_gpu[i]/*+radius*n1*n2*/, img+i*n1*n2*(n3-2*radius), n1*n2*(n3/*-2*radius*/)*sizeof(float), hipMemcpyHostToDevice);
  }

  for(int i=0; i<n_gpus; i++){
    hipSetDevice(i);
    hipFree(data_p0[i]);
    hipFree(img_gpu[i]);
    hipFree(data_p1[i]);
    hipFree(src_p0[i]);
    hipFree(src_p1[i]);
  }
  hipFree(data_gpu);
  hipFree(datageom_gpu);
  hipFree(source_gpu0);
  hipFree(srcgeom_gpu0);
}
void transfer_sinc_table_s(int nsinc, int ns,  float **tables){
	hipSetDevice(0);
   float *tmp_table1=(float*)malloc(sizeof(float)*nsinc*ns);
   for(int i=0; i < ns; i++) memcpy((tmp_table1+nsinc*i),tables[i],nsinc*sizeof(float));
   hipMalloc((void**) &sincstable,ns*nsinc*sizeof(float));
   hipMemcpy(sincstable, tmp_table1, ns*nsinc*sizeof(float),hipMemcpyHostToDevice);
   hipMemcpyToSymbol(HIP_SYMBOL(sinc_s_table), &sincstable, sizeof(float*));
   free(tmp_table1);
   hipMemcpyToSymbol(HIP_SYMBOL(nsinc_gpu), &nsinc, sizeof(int));
 }
void transfer_sinc_table_d(int nsinc,  int nd, float **tabled){
	hipSetDevice(0);
   float *tmp_table2=(float*)malloc(sizeof(float)*nsinc*nd);
   for(int i=0; i < nd; i++) memcpy((tmp_table2+nsinc*i),tabled[i],nsinc*sizeof(float));
   hipMalloc((void**) &sincdtable,nd*nsinc*sizeof(float));
   hipMemcpy(sincdtable, tmp_table2, nd*nsinc*sizeof(float),hipMemcpyHostToDevice);
   hipMemcpyToSymbol(HIP_SYMBOL(sinc_d_table), &sincdtable, sizeof(float*));
   free(tmp_table2);
}
void load_source(int it){
  int ibeg=it;
  int iend=it+ntblock_internal+7;
  int ntb=ntblock_internal+7;
  if(iend> ntsource_internal) iend=ntsource_internal;
  float *vals=(float*)malloc(ntb*sizeof(float)*npts_internal);
  float sm=0;
  for(int ipt=0; ipt< npts_internal; ipt++){
    memcpy(&vals[ipt*ntb],
      &source_buf[it+ntsource_internal*ipt],(iend-ibeg)*sizeof(float));
      for(int i=0; i < iend-ibeg; i++){
      sm+=fabs(vals[i+(iend-ibeg)*ipt]);
      }
      if(iend-ibeg< ntblock_internal+7){ 
       for(int it=0; it< ntblock_internal-7-(iend-ibeg); it++) {
         vals[ipt*ntb+(iend-ibeg)+it]=0.;
      }
     }
  }
  fprintf(stderr,"SM LOAD SOUrCE %f \n",sm);
  hipMemcpy(source_gpu, vals,ntb*npts_internal*sizeof(float), hipMemcpyHostToDevice);
  free(vals);
}
void set_ntblock(int nblock){
      hipSetDevice(device[0]);
   ntblock_internal=nblock;
   int nt=ntblock_internal+7;
  hipMemcpyToSymbol(HIP_SYMBOL(ntblock_gpu), &nt, sizeof(int));
}
void transfer_source_func(int npts, int nt, int *locs, float *vals){
   shot_gpu=0;
   hipSetDevice(device[0]);
   npts_internal=npts;
   ntsource_internal=nt;
   hipMalloc((void**) &source_gpu,nt*npts*sizeof(float));
   hipMalloc((void**) &srcgeom_gpu,npts*sizeof(int));
   hipMemset(source_gpu, 0,(nt*npts)*sizeof(float));
   hipMemcpy(srcgeom_gpu, locs, npts*sizeof(int), hipMemcpyHostToDevice);
int imin,imax=imin=locs[0];
for(int i=0;i < npts; i++){
  if(locs[i]<imin) imin=locs[i];
  if(locs[i]>imax) imax=locs[i];

}
   source_buf=(float*)malloc(sizeof(float)*nt*npts);
   memcpy(source_buf,vals,nt*npts*sizeof(float));
float sm=0;
fprintf(stderr,"CHECK MIN MAX %d %d \n",imin,imax);
for(int i=0;i < nt*npts; i++){
 sm+=fabsf(source_buf[i]);
 }
 fprintf(stderr,"SM %f \n",sm);

  // srite("srccheck.H",vals,nt*npts*4);
  // fprintf(stderr,"%d %d \n",nt,npts);
  // hipMemcpy(source_gpu, vals, nt*npts*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpyToSymbol(HIP_SYMBOL(ntrace_gpu), &nt, sizeof(int));
   hipMemcpyToSymbol(HIP_SYMBOL(srcgeom_gpu0), &srcgeom_gpu, sizeof(int*));
      hipMemcpyToSymbol(HIP_SYMBOL(npts_gpu), &npts, sizeof(int));

  hipMemcpyToSymbol(HIP_SYMBOL(source_gpu0), &source_gpu, sizeof(float*));
//         int ntt=ntblock_internal+7;
//  hipMemcpyToSymbol(HIP_SYMBOL(ntblock_gpu), &ntt, sizeof(int));
}
void transfer_receiver_func(int nx, int ny, int nt, int *locs, float *vals){
   hipSetDevice(device[0]);
  // hipMalloc((void**) &data_gpu,nt*nx*ny*sizeof(float));
   hipMalloc((void**) &data_gpu,(7+ntblock_internal)*nx*ny*sizeof(float));
      hipMemset(data_gpu, 0,(7+ntblock_internal)*nx*ny*sizeof(float));
  hipMalloc((void**) &datageom_gpu,nx*ny*sizeof(float));

fprintf(stderr,"TRASN RECEIVER %d %d %d \n",nx,ny,nx*ny);
   hipMemcpy(datageom_gpu, locs, nx*ny*sizeof(float),	 hipMemcpyHostToDevice);
  //hipMemcpy(data_gpu, vals,(7+ ntblock_internal)*nx*ny*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(rec_nx_gpu), &nx, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(rec_ny_gpu), &ny, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(ntrace_gpu), &nt, sizeof(int));
   hipMemcpyToSymbol(HIP_SYMBOL(datageom_gpu0), &datageom_gpu, sizeof(int*));
   hipMemcpyToSymbol(HIP_SYMBOL(data_gpu0), &data_gpu, sizeof(float*));
 //     int ntt=ntblock_internal+7;
 // hipMemcpyToSymbol(HIP_SYMBOL(ntblock_gpu), &ntt, sizeof(int));
}
void transfer_vel_func1(int n1, int n2, int n3, float *vel){
    n3=(n3-2*radius)/n_gpus+2*radius;
    for(int i=0; i<n_gpus; i++){
      hipSetDevice(device[i]);
      hipMemcpy( velocity[i]/*+lead_pad*/, vel+i*n1*n2*(n3-2*radius), n1*n2*n3*sizeof(float), hipMemcpyHostToDevice);
    }
}
void transfer_vel_func2(int n1, int n2, int n3, float *vel){
    n3=(n3-2*radius)/n_gpus+2*radius;
    for(int i=0; i<n_gpus; i++){
      hipSetDevice(device[i]);
      hipMemcpy( velocity2[i]/*+lead_pad*/, vel+i*n1*n2*(n3-2*radius), n1*n2*n3*sizeof(float), hipMemcpyHostToDevice);
    }
}
void create_gpu_space(float d1, float d2, float d3, float bc_a, float bc_b, float bc_b_y, int n1, int n2, int n3){

    lead_pad=0;//32-radius;
    n3=(n3-2*radius)/n_gpus+2*radius;
    offset=radius*n1*n2+radius*n1+radius+lead_pad;

    float coeffs_cpu[COEFFS_SIZE]=get_coeffs((double)d1,(double)d2,(double)d3);

    dd1=1./(double)d1/(double)d1;
    dd2=1./(double)d2/(double)d2;
    dd3=1./(double)d3/(double)d3;

    for(int i=0; i<n_gpus; i++){
      hipSetDevice(device[i]);
      hipMalloc( (void**)&velocity[i], (n1*n2*n3/*+lead_pad*/)*sizeof(float));
      hipMalloc( (void**)&velocity2[i], (n1*n2*n3/*+lead_pad*/)*sizeof(float));
    fprintf(stderr,"CHECK N1GPU %d \n",n1);
      hipMemcpyToSymbol(HIP_SYMBOL(n1gpu), &n1, sizeof(int));
      hipMemcpyToSymbol(HIP_SYMBOL(n2gpu), &n2, sizeof(int));
      hipMemcpyToSymbol(HIP_SYMBOL(n3gpu), &n3, sizeof(int));

      hipMemcpyToSymbol(HIP_SYMBOL(bc_agpu), &bc_a, sizeof(float));
      hipMemcpyToSymbol(HIP_SYMBOL(bc_bgpu), &bc_b, sizeof(float));
      hipMemcpyToSymbol(HIP_SYMBOL(coeffs), coeffs_cpu, COEFFS_SIZE*sizeof(float), 0,hipMemcpyHostToDevice);

    }
}

